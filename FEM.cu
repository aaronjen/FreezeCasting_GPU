#include "hip/hip_runtime.h"
#include "FEM.h"
#include <Eigen/Dense>
#include <Eigen/Sparse>
#include <cmath>
#include <vector>
#include <memory>
#include <fstream>
#include <ctime>
#include "GaussPoints.h"
#include "ShapeFunctions.h"
#include "Quadtree.h"
using namespace std;
using namespace Eigen;

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

#define BLOCK_SIZE 256
#define BLOCK_WARP 8




void FEM::MeshRefinement() {
	PhiCoordinateList.clear();
	UCoordinateList.clear();
	PhiVelocityCoordinateList.clear();
	UVelocityCoordinateList.clear();
	for (int i = 0; i < PHI.rows(); i++) {
		PhiCoordinateList[NodeCoordinates[i]] = PHI(i);
		UCoordinateList[NodeCoordinates[i]] = U(i);
		PhiVelocityCoordinateList[NodeCoordinates[i]] = PHIvelocity(i);
		UVelocityCoordinateList[NodeCoordinates[i]] = Uvelocity(i);
	}

	Quadtree_MeshGenerate(maxLv, gamma, LevelElementList, 10, PhiCoordinateList, UCoordinateList, PhiVelocityCoordinateList, UVelocityCoordinateList); // case = 10
	Quadtree_AddNodes(LevelElementList, NodeCoordinateList);
	ReportElement(LevelElementList, FinalElementList, NodeCoordinateList, EFT, NodeCoordinates);

	ncSize = NodeCoordinates.size();

	PHI.setZero(ncSize);
	U.setZero(ncSize);
	Theta.setZero(ncSize);
	PHIvelocity.setZero(ncSize);
	Uvelocity.setZero(ncSize);
	for (unsigned i = 0; i < ncSize; i++) {
		PHI(i) = PhiCoordinateList[NodeCoordinates[i]];
		U(i) = UCoordinateList[NodeCoordinates[i]];
		Theta(i) = 0;
		PHIvelocity(i) = PhiVelocityCoordinateList[NodeCoordinates[i]];
		Uvelocity(i) = UVelocityCoordinateList[NodeCoordinates[i]];
	}
	//fout_time << endl;
	//fout_time << EFT.size() << "\tElements" << endl;
	//fout_time << NodeCoordinateList.size() << "\tNodes" << endl;
	//fout_time << endl;

	// const VectorXd& PHI,
	// const VectorXd& U,
	// const vector<vector<int>>& EFT,
	// const vector<shared_ptr<Element>>& FinalElementList,
	// const vector<Coord>& NodeCoordinates,

	hipFree(aPHI);
	hipFree(aU);

	hipMalloc(&aPHI, sizeof(double)*ncSize);
	hipMalloc(&aU, sizeof(double)*ncSize);
	

	// cuda pointer
	hipFree(aM11);
	hipFree(aM21);
	hipFree(aM22);
	hipFree(aK11);
	hipFree(aK21);
	hipFree(aK22);
	hipFree(aF1);

	hipMallocManaged(&aM11, sizeof(double)*ncSize*ncSize);
	hipMallocManaged(&aM21, sizeof(double)*ncSize*ncSize);
	hipMallocManaged(&aM22, sizeof(double)*ncSize*ncSize);
	hipMallocManaged(&aK11, sizeof(double)*ncSize*ncSize);
	hipMallocManaged(&aK21, sizeof(double)*ncSize*ncSize);
	hipMallocManaged(&aK22, sizeof(double)*ncSize*ncSize);
	hipMallocManaged(&aF1, sizeof(double)*ncSize);
}


void FEM::find_matrixs(double lambda, double epsilon, unsigned tloop, double dt) {
	// initialization
	const double PI = 3.14159265358979323846;
	double C_inf = 3;	// (wt%)
	double k = 0.14;		// 
	double G = 140 * 40;		// (K/cm)
	double d0 = 5E-3;		// (1E-6m)
	double alpha = 3000;	// (1E-6m2/s)
	double Vp = 3000;		// (1E-6m/s)
	double Ts = 273;		// (K)
	double dT0 = 2.6 * C_inf * (1 - k) / k;		// (K)
	double T0 = Ts - dT0 / 10;		// (K)
	double a1 = 0.8839;
	double a2 = 0.6267;
	double W0 = d0 * lambda / a1; // (1E-6m)
	double Tau0 = a2 * lambda * W0 * W0 / alpha; // (s)
	double D = lambda * a2;

	
	Map<MatrixXd> mM11(aM11, ncSize, ncSize);
	Map<MatrixXd> mM21(aM21, ncSize, ncSize);
	Map<MatrixXd> mM22(aM22, ncSize, ncSize);
	Map<MatrixXd> mK11(aK11, ncSize, ncSize);
	Map<MatrixXd> mK21(aK21, ncSize, ncSize);
	Map<MatrixXd> mK22(aK22, ncSize, ncSize);
	Map<VectorXd> vF1(aF1, ncSize);

	mM11 = MatrixXd::Zero(ncSize, ncSize);
	mM21 = MatrixXd::Zero(ncSize, ncSize);
	mM22 = MatrixXd::Zero(ncSize, ncSize);
	mK11 = MatrixXd::Zero(ncSize, ncSize);
	mK21 = MatrixXd::Zero(ncSize, ncSize);
	mK22 = MatrixXd::Zero(ncSize, ncSize);
	vF1  = VectorXd::Zero(ncSize);

	int nGp = 2 * 2; // 2 x 2 Gauss point
	MatrixXd LocationsAndWeights = gauss2D(nGp);
	int m = 6;
	double RealTime = Tau0 * tloop; // (s)
	
	for (unsigned e = 0; e < EFT.size(); e++) {
		size_t numNodePerElement = EFT[e].size();
		bitset<8> bitElementType = FinalElementList[e]->bitElementType;
		// get the coordinates of the nodes in the element
		MatrixXd elementNodesCoord(numNodePerElement,2); // n x 2
		VectorXd phi(numNodePerElement); // n x 1
		VectorXd u(numNodePerElement); // n x 1

		// element info
		for (unsigned i = 0; i < numNodePerElement; i++) {
			int nodeSerial = EFT[e][i];

			elementNodesCoord(i, 0) = NodeCoordinates[nodeSerial].x;
			elementNodesCoord(i, 1) = NodeCoordinates[nodeSerial].y;
			phi(i) = PHI[nodeSerial];
			u(i) = U[nodeSerial];
        }
		double RealCoord = W0 * 0.25*(elementNodesCoord(0, 1) + elementNodesCoord(1, 1) + elementNodesCoord(2, 1) + elementNodesCoord(3, 1)) * 1E-6; // n x 2 (m)
		
		MatrixXd Ce = MatrixXd::Zero(numNodePerElement, numNodePerElement);
		MatrixXd Ae = MatrixXd::Zero(numNodePerElement, numNodePerElement);
		MatrixXd Ee = MatrixXd::Zero(numNodePerElement, numNodePerElement);
		VectorXd Fe = VectorXd::Zero(numNodePerElement); // n x 1

		RowVectorXd N0 = ShapeFunction(0, 0, bitElementType);
		MatrixXd dN0 = NaturalDerivatives(0, 0, bitElementType); // 2 x n
		MatrixXd B0 = XYDerivatives(elementNodesCoord, dN0); // 2 x n
		MatrixXd cotangent = get_cotangent(phi, B0); // 2 x 1
		double DERX = cotangent(0);
		double DERY = cotangent(1);
		double angle = atan2(DERY, DERX);
		double as = 1 + epsilon * cos(m*(angle - PI / 6)); // A(theta)
		double asp = -m * epsilon * sin(m*(angle - PI / 6)); // A'(theta)
		double Temperature = T0 + G * 1E2 * (W0 * N0 * elementNodesCoord.col(1) - Vp*RealTime) * 1E-6; // (K)
		double theta = (Temperature - Ts) / dT0;

		// cycle for Gauss point
		for (int q=0; q<nGp; q++) {
			double xi = LocationsAndWeights(q,0);
			double eta = LocationsAndWeights(q,1);
			double W = LocationsAndWeights(q,2);
			RowVectorXd N = ShapeFunction(xi, eta, bitElementType); // 1 x n
			MatrixXd dN = NaturalDerivatives(xi, eta, bitElementType); // 2 x n
			MatrixXd B = XYDerivatives(elementNodesCoord, dN); // 2 x n
			double J = detJacobian(elementNodesCoord, dN); // 1 x 1
			// matrixs of a element
			Ce     += N.transpose() * N * W * J; // n x n
			Ae     -= B.transpose() * B * W * J; // n x n
			Ee	   -= (B.row(1).transpose()*B.row(0) - B.row(0).transpose()*B.row(1)) * W * J; // n x n
			Fe	   += N.transpose() * f(N*phi, N*u, theta, lambda) * W * J; // n x 1
		}

		for (unsigned i=0; i<numNodePerElement; i++) {
			int x = EFT[e][i];
			for (unsigned j=0; j<numNodePerElement; j++) {
				int y = EFT[e][j];
				if (Ce(i, j) > 1.0E-12 || Ce(i, j) < -1.0E-12) {
					mM22(x, y) += Ce(i, j);
					mM21(x, y) += -0.5*Ce(i, j);
					mM11(x, y) += as * as * Ce(i, j);
				}
				if (Ae(i, j) > 1.0E-12 || Ae(i, j) < -1.0E-12) {
					mK22(x, y) += -D * q(N0 * phi, 0.7) * Ae(i, j);
					mK11(x, y) += -as * as * Ae(i, j);
				}
				if (Ee(i, j) > 1.0E-12 || Ee(i, j) < -1.0E-12)
					mK11(x, y) += -as * asp * Ee(i, j);
			}
			if (Fe(i) > 1.0E-12 || Fe(i) < -1.0E-12)
				vF1(x) += Fe(i);
        	}
	}
}

// __global__ void cu_element(
// 	double lambda,
// 	double tloop,
// 	double epsilon,
// 	const VectorXd& PHI,
// 	const VectorXd& U,
// 	const vector<vector<int>>& EFT,
// 	const vector<shared_ptr<Element>>& FinalElementList,
// 	const vector<Coord>& NodeCoordinates,
// 	double* aM11,
// 	double* aM21,
// 	double* aM22,
// 	double* aK11,
// 	double* aK21,
// 	double* aK22,
// 	double* aF1,
// 	int ncSize
// ){
// 	const double PI = 3.14159265358979323846;
// 	double C_inf = 3;	// (wt%)
// 	double k = 0.14;		// 
// 	double G = 140 * 40;		// (K/cm)
// 	double d0 = 5E-3;		// (1E-6m)
// 	double alpha = 3000;	// (1E-6m2/s)
// 	double Vp = 3000;		// (1E-6m/s)
// 	double Ts = 273;		// (K)
// 	double dT0 = 2.6 * C_inf * (1 - k) / k;		// (K)
// 	double T0 = Ts - dT0 / 10;		// (K)
// 	double a1 = 0.8839;
// 	double a2 = 0.6267;
// 	double W0 = d0 * lambda / a1; // (1E-6m)
// 	double Tau0 = a2 * lambda * W0 * W0 / alpha; // (s)
// 	double D = lambda * a2;

// 	Map<MatrixXd> mM11(aM11, ncSize, ncSize);
// 	Map<MatrixXd> mM21(aM21, ncSize, ncSize);
// 	Map<MatrixXd> mM22(aM22, ncSize, ncSize);
// 	Map<MatrixXd> mK11(aK11, ncSize, ncSize);
// 	Map<MatrixXd> mK21(aK21, ncSize, ncSize);
// 	Map<MatrixXd> mK22(aK22, ncSize, ncSize);
// 	Map<VectorXd> vF1(aF1, ncSize);


// 	int e = blockIdx.x * blockDim.x + threadIdx.x;
// 	if (e < EFT.size()){
// 		const double PI = 3.14159265358979323846;
//         double C_inf = 3;       // (wt%)
//         double k = 0.14;                // 
//         double G = 140 * 40;            // (K/cm)
//         double d0 = 5E-3;               // (1E-6m)
//         double alpha = 3000;    // (1E-6m2/s)
//         double Vp = 3000;               // (1E-6m/s)
//         double Ts = 273;                // (K)
//         double dT0 = 2.6 * C_inf * (1 - k) / k;         // (K)
//         double T0 = Ts - dT0 / 10;              // (K)
//         double a1 = 0.8839;
//         double a2 = 0.6267;
//         double W0 = d0 * lambda / a1; // (1E-6m)
//         double Tau0 = a2 * lambda * W0 * W0 / alpha; // (s)
//         double D = lambda * a2;
// 		int nGp = 2 * 2; // 2 x 2 Gauss point
// 		MatrixXd LocationsAndWeights = gauss2D(nGp);
// 		int m = 6;
//         double RealTime = Tau0 * tloop; // (s)

//         size_t numNodePerElement = EFT[e].size();
//         bitset<8> bitElementType = FinalElementList[e]->bitElementType;
//         MatrixXd elementNodesCoord(numNodePerElement,2);
//         VectorXd phi(numNodePerElement);
//         VectorXd u(numNodePerElement);

//         for (unsigned i = 0; i < numNodePerElement; i++) {
//         	int nodeSerial = EFT[e][i];
//         	elementNodesCoord(i, 0) = NodeCoordinates[nodeSerial].x;
//         	elementNodesCoord(i, 1) = NodeCoordinates[nodeSerial].y;
//         	phi(i) = PHI[nodeSerial];
//         	u(i) = U[nodeSerial];
//         }
//         double RealCoord = W0 * 0.25*(elementNodesCoord(0, 1) + elementNodesCoord(1, 1) + elementNodesCoord(2, 1) + elementNodesCoord(3, 1)) * 1E-6;

//         MatrixXd Ce = MatrixXd::Zero(numNodePerElement, numNodePerElement);
//         MatrixXd Ae = MatrixXd::Zero(numNodePerElement, numNodePerElement);
//         MatrixXd Ee = MatrixXd::Zero(numNodePerElement, numNodePerElement);
// 		VectorXd Fe = VectorXd::Zero(numNodePerElement); // n x 1
// 		RowVectorXd N0 = ShapeFunction(0, 0, bitElementType);
// 		MatrixXd dN0 = NaturalDerivatives(0, 0, bitElementType); // 2 x n
// 		MatrixXd B0 = XYDerivatives(elementNodesCoord, dN0); // 2 x n
// 		MatrixXd cotangent = get_cotangent(phi, B0); // 2 x 1
// 		double DERX = cotangent(0);
// 		double DERY = cotangent(1);
// 		double angle = atan2(DERY, DERX);
// 		double as = 1 + epsilon * cos(m*(angle - PI / 6)); // A(theta)
// 		double asp = -m * epsilon * sin(m*(angle - PI / 6)); // A'(theta)
// 		double Temperature = T0 + G * 1E2 * (W0 * N0 * elementNodesCoord.col(1) - Vp*RealTime) * 1E-6; // (K)
// 		double theta = (Temperature - Ts) / dT0;

// 		for (int q=0; q<nGp; q++) {
// 			double xi = LocationsAndWeights(q,0);
// 			double eta = LocationsAndWeights(q,1);
// 			double W = LocationsAndWeights(q,2);
// 			RowVectorXd N = ShapeFunction(xi, eta, bitElementType); // 1 x n
// 			MatrixXd dN = NaturalDerivatives(xi, eta, bitElementType); // 2 x n
// 			MatrixXd B = XYDerivatives(elementNodesCoord, dN); // 2 x n
// 			double J = detJacobian(elementNodesCoord, dN); // 1 x 1
// 	                // matrixs of a element
// 			Ce     += N.transpose() * N * W * J; // n x n
// 			Ae     -= B.transpose() * B * W * J; // n x n
// 			Ee         -= (B.row(1).transpose()*B.row(0) - B.row(0).transpose()*B.row(1)) * W * J; // n x n
// 			Fe         += N.transpose() * f(N*phi, N*u, theta, lambda) * W * J; // n x 1
// 		}
		
// 		for (unsigned i=0; i<numNodePerElement; i++) {
// 			int x = EFT[e][i];
// 			for (unsigned j=0; j<numNodePerElement; j++) {
// 				int y = EFT[e][j];
// 				if (Ce(i, j) > 1.0E-12 || Ce(i, j) < -1.0E-12) {
// 					mM22(x, y) += Ce(i, j);
// 					mM21(x, y) += -0.5*Ce(i, j);
// 					mM11(x, y) += as * as * Ce(i, j);
// 				}
// 				if (Ae(i, j) > 1.0E-12 || Ae(i, j) < -1.0E-12) {
// 					mK22(x, y) += -D * q(N0 * phi, 0.7) * Ae(i, j);
// 					mK11(x, y) += -as * as * Ae(i, j);
// 				}
// 				if (Ee(i, j) > 1.0E-12 || Ee(i, j) < -1.0E-12)
// 					mK11(x, y) += -as * asp * Ee(i, j);
// 			}
// 			if (Fe(i) > 1.0E-12 || Fe(i) < -1.0E-12)
// 				vF1(x) += Fe(i);
// 		}
// 	}
// }

void FEM::cu_find_matrixs(double lambda, double epsilon, unsigned tloop, double dt){

	hipMemcpy(aPHI, PHI.data(), sizeof(double)*ncSize, hipMemcpyHostToDevice);
	hipMemcpy(aU, U.data(), sizeof(double)*ncSize, hipMemcpyHostToDevice);


}



void FEM::time_discretization(
	double lambda,
	double epsilon,
	unsigned tloop,
	double dt) {
	clock_t t;
	clock_t solver_time = 0;
	clock_t matrix_time = 0;
	clock_t scheme_time = 0;

	t = clock(); //-> solver
	BiCGSTAB<SparseMatrix<double> > solver;
	solver_time += clock() - t; //<- solver

	///////////////////////////////////////////////////////////////////////////////////////////////////
	t = clock(); //-> scheme
	double rho = 0;
	double rhos = 0;
	double W1L4 = 1 / (1 + rho);
	double W1L6 = (3 + rho + rhos - rho*rhos) / (2 * (1 + rho) * (1 + rhos));
	double lambda4 = 1;
	double lambda5 = 1 / (1 + rhos);
	unsigned nNode = ncSize;

	typedef Triplet<double> T;
	vector<T> tripletList_q;
	vector<T> tripletList_Up, tripletList_Down, tripletList_Left, tripletList_Right;
	for (unsigned i = 0; i < nNode; i++) {
		tripletList_Up.push_back(T(i, i, 1));
		tripletList_Down.push_back(T(i + nNode, i, 1));
		tripletList_Left.push_back(T(i, i, 1));
		tripletList_Right.push_back(T(i, i + nNode, 1));
	}

	SparseMatrix<double> Up(nNode * 2, nNode); 
	Up.setFromTriplets(tripletList_Up.begin(), tripletList_Up.end());
	SparseMatrix<double> Down(nNode * 2, nNode);
	Down.setFromTriplets(tripletList_Down.begin(), tripletList_Down.end());
	SparseMatrix<double> Left(nNode, nNode * 2);
	Left.setFromTriplets(tripletList_Left.begin(), tripletList_Left.end());
	SparseMatrix<double> Right(nNode, nNode * 2);
	Right.setFromTriplets(tripletList_Right.begin(), tripletList_Right.end());


	VectorXd d1 = Up * PHI + Down * U;
	VectorXd v1;
	if (tloop == 0) {
		PHIvelocity *= 0;
		find_matrixs(lambda, epsilon, tloop, dt);

		SparseMatrix<double> mM11 = Map<MatrixXd>(aM11, ncSize, ncSize).sparseView();
		SparseMatrix<double> mM21 = Map<MatrixXd>(aM21, ncSize, ncSize).sparseView();
		SparseMatrix<double> mM22 = Map<MatrixXd>(aM22, ncSize, ncSize).sparseView();
		SparseMatrix<double> mK11 = Map<MatrixXd>(aK11, ncSize, ncSize).sparseView();
		SparseMatrix<double> mK21 = Map<MatrixXd>(aK21, ncSize, ncSize).sparseView();
		SparseMatrix<double> mK22 = Map<MatrixXd>(aK22, ncSize, ncSize).sparseView();
		Map<VectorXd> vF1(aF1, ncSize);

		SparseMatrix<double> M = Up*(mM11)*Left + Down*(mM21)*Left + Down*(mM22)*Right;
		SparseMatrix<double> K = Up*(mK11)*Left + Down*(mK21)*Left + Down*(mK22)*Right;
		VectorXd F = Up * vF1;
		v1 = solver.compute(M).solve(F - K*d1);
	} else {
		v1 = Up * PHIvelocity + Down * Uvelocity;
	}

	VectorXd d_telda = d1 + W1L4 * v1 * dt;
	PHI = d_telda.topRows(nNode);
	U = d_telda.bottomRows(nNode);
	scheme_time += clock() - t; //<- scheme
	
	t = clock(); //-> matrix

	// cu_find_matrixs(lambda, epsilon, tloop, dt);

	find_matrixs(lambda, epsilon, tloop, dt);
	SparseMatrix<double> mM11 = Map<MatrixXd>(aM11, ncSize, ncSize).sparseView();
	SparseMatrix<double> mM21 = Map<MatrixXd>(aM21, ncSize, ncSize).sparseView();
	SparseMatrix<double> mM22 = Map<MatrixXd>(aM22, ncSize, ncSize).sparseView();
	SparseMatrix<double> mK11 = Map<MatrixXd>(aK11, ncSize, ncSize).sparseView();
	SparseMatrix<double> mK21 = Map<MatrixXd>(aK21, ncSize, ncSize).sparseView();
	SparseMatrix<double> mK22 = Map<MatrixXd>(aK22, ncSize, ncSize).sparseView();
	Map<VectorXd> vF1(aF1, ncSize);

	matrix_time += clock() - t;	 //<- matrix

	t = clock(); //-> scheme
	SparseMatrix<double> M = Up*(mM11)*Left + Down*(mM21)*Left + Down*(mM22)*Right;
	SparseMatrix<double> K = Up*(mK11)*Left + Down*(mK21)*Left + Down*(mK22)*Right;
	VectorXd F = Up * vF1;
	scheme_time += clock() - t; //<- scheme

	t = clock(); //-> solver
	VectorXd v_telda = solver.compute( M ).solve( F - K*d_telda );
	solver_time += clock() - t; //<- solver

	t = clock(); //-> scheme
	VectorXd dv = (-v1 + v_telda) / W1L6;
	VectorXd d2 = d1 + lambda4 * v1 * dt + lambda5 * dv * dt;
	VectorXd v2 = v1 + dv;

	PHI = d2.topRows(nNode);
	U = d2.bottomRows(nNode);
	PHIvelocity = v2.topRows(nNode);
	Uvelocity = v2.bottomRows(nNode);
	scheme_time += clock() - t; //<- scheme

	//fout_time << "\tmatrix: " << 1.*matrix_time/CLOCKS_PER_SEC << " sec" << endl;
	//fout_time << "\tsolver: " << 1.*solver_time/CLOCKS_PER_SEC << " sec" << endl;
	//fout_time << "\tscheme: " << 1.*scheme_time/CLOCKS_PER_SEC << " sec" << endl;
	//cout << "\tmatrix: " << 1.*matrix_time/CLOCKS_PER_SEC << " sec" << endl;
	//cout << "\tsolver: " << 1.*solver_time/CLOCKS_PER_SEC << " sec" << endl;
	//cout << "\tscheme: " << 1.*scheme_time/CLOCKS_PER_SEC << " sec" << endl;
	
}

__device__ __host__ MatrixXd get_cotangent(const VectorXd& phi, const MatrixXd& B) {
////////////////////////////////////////////////////////////////////////
// phi = / phi1 \     B =  / N1,x N2,x N3,x N4,x \     cot = / DERX \ //
//       | phi2 |          \ N1,y N2,y N3,y N4,y /           \ DERY / //
//       | phi3 |                                                     //
//       \ phi4 /                                                     //
////////////////////////////////////////////////////////////////////////
    return B * phi; // 2x1
}

// g'(phi) - lambda*U*P'(phi)
__device__ __host__ double f(double phi, double u, double theta, double lambda) {
	return phi * (1 - phi*phi) - lambda * u * pow(1 - phi*phi, 2.0);
	//return phi * (1 - phi*phi) - lambda * pow(1 - phi*phi, 2.0) * (u + 0.9 * phi * (1 - phi*phi) * ((double(rand()) / RAND_MAX) - 0.5));
	//return phi * (1 - phi*phi) - lambda * pow((1 - phi*phi), 2.0) * (u + theta);
	//return phi * (1 - phi*phi) - lambda * pow((1 - phi*phi), 2.0) * (u + theta + 0.3 * phi * (1 - phi*phi) * ((double(rand()) / RAND_MAX) - 0.5));
}

__device__ __host__ double q(double phi, double k) {
	//return (phi >= 1) ? 0 : (1 - phi) / (1 + k - (1 - k) * phi);
	return (phi >= 1) ? 0 : (1 - phi) / (1 + k - (1 - k) * phi) + (1 + phi) * 0.2 / 2;
	//return (phi >= 1) ? 0 : (1 - phi) / 2;
	//return (phi >= 1) ? 0 : (1 - phi) / 2 + (1 + phi) * 0.2 / 2;
}
